#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include "util.h"

// The width and height of a sudoku board
#define BOARD_DIM 9

// The width and heigh of a square group in a sudoku board
#define GROUP_DIM 3

// The number of boards to pass to the solver at one time
#define BATCH_SIZE 25000

/**
 * A board is an array of 81 cells. Each cell is encoded as a 16-bit integer.
  */
typedef struct board {
  uint16_t cells[BOARD_DIM * BOARD_DIM];
} board_t;

// Declare a few functions. 
void print_board(board_t* board);
__host__ __device__ uint16_t digit_to_cell(int digit);
__host__ __device__ int cell_to_digit(uint16_t cell);

/**
 * This is the kernal to solve the sudoku boards in GPU.
 * more than BATCH_SIZE, but may be less if the total number of input
 *
 *
 * \param boards      An array of boards that should be solved.
 */
__global__ void cell_solver(board_t* boards) {
  size_t cell_idx = threadIdx.x;
  uint16_t current_cell;
  size_t votes;

  // shared memory for all the threads in the block.
  __shared__ board_t board;
  // copy the contents of the board into the shared memory
  board.cells[cell_idx] = boards[blockIdx.x].cells[cell_idx];
  // wait for all the threads to finish copying the boards.
  __syncthreads();

  do {
    current_cell = board.cells[cell_idx];
    if (cell_to_digit(current_cell) != 0) break;
    // loop through the col
    size_t col_idx = cell_idx % 9;
    for (size_t index = col_idx; index < col_idx + 9 * 9; index += 9) {
      if (index == cell_idx) continue;
      int digit_result = cell_to_digit(board.cells[index]);
      if (digit_result != 0) board.cells[cell_idx] &= ~(1 << digit_result);
    }
    if (cell_to_digit(current_cell) != 0) break;
    // loop through the row
    size_t start_idx = cell_idx - col_idx;
    for (size_t index = start_idx; index < start_idx + 9; index++) {
      if (index == cell_idx) continue;
      int digit_result = cell_to_digit(board.cells[index]);
      if (digit_result != 0) board.cells[cell_idx] &= ~(1 << digit_result);
    }
    if (cell_to_digit(current_cell) != 0) break;
    // find the index of the top left corner of the square
    // reduced_index is the index of cell that has the same column
    // index but is in the first row.
    size_t reduced_index = cell_idx - (cell_idx / 27) * 27;
    size_t minor_row = reduced_index / 9;
    size_t minor_col = (reduced_index - minor_row * 9) % 3;
    // start_index is the index of cell at the top left corner that
    // share the same square of the current cell.
    size_t start_index = cell_idx - minor_col - minor_row * 9;
    // loop through the square
    for (size_t row = 0; row < 3; row++) {
      for (size_t col = 0; col < 3; col++) {
        size_t index = start_index + col + row * 9;
        if (index == cell_idx) continue;
        int digit_result = cell_to_digit(board.cells[index]);
        if (digit_result != 0) board.cells[cell_idx] &= ~(1 << digit_result);
      }
    }
    votes = __syncthreads_count(board.cells[cell_idx] != current_cell);

  } while (votes != 0);

  boards[blockIdx.x].cells[cell_idx] = board.cells[cell_idx];
}

/**
 * Take an array of boards and solve them all. 
 *
 * \param boards      An array of boards that should be solved.
 * \param num_boards  The numebr of boards in the boards array
 */
void solve_boards(board_t* cpu_boards, size_t num_boards) {
  // allocate memory in gpu
  board_t* gpu_boards;
  if (hipMalloc(&gpu_boards, sizeof(board_t) * num_boards) != hipSuccess) {
    perror("cuda malloc failed.");
    exit(2);
  }
  // copy the content over to gpu
  if (hipMemcpy(gpu_boards, cpu_boards, sizeof(board_t) * num_boards, hipMemcpyHostToDevice) !=
      hipSuccess) {
    perror("cuda memcpy failed. ");
    exit(2);
  }
  // run the kernal over BATCH_SIZE blocks and 81 threads
  cell_solver<<<BATCH_SIZE, 81>>>(gpu_boards);
  // wait for all the threads to finish
  if (hipDeviceSynchronize() != hipSuccess) {
    perror("Synchronized failed.");
    exit(2);
  }
  // copy contents from gpu to cpu.
  if (hipMemcpy(cpu_boards, gpu_boards, sizeof(board_t) * num_boards, hipMemcpyDeviceToHost) !=
      hipSuccess) {
    perror("cuda memcpy failed. ");
    exit(2);
  }
}

/**
 * Take as input an integer value 0-9 (inclusive) and convert it to the encoded
 * cell form used for solving the sudoku. This encoding uses bits 1-9 to
 * indicate which values may appear in this cell.
 *
 * For example, if bit 3 is set to 1, then the cell may hold a three. Cells that
 * have multiple possible values will have multiple bits set.
 *
 * The input digit 0 is treated specially. This value indicates a blank cell,
 * where any value from one to nine is possible.
 *
 * \param digit   An integer value 0-9 inclusive
 * \returns       The encoded form of digit using bits to indicate which values
 *                may appear in this cell.
 */
__host__ __device__ uint16_t digit_to_cell(int digit) {
  if (digit == 0) {
    // A zero indicates a blank cell. Numbers 1-9 are possible, so set bits 1-9.
    return 0x3FE;
  } else {
    // Otherwise we have a fixed value. Set the corresponding bit in the board.
    return 1 << digit;
  }
}

/*
 * Convert an encoded cell back to its digit form. A cell with two or more
 * possible values will be encoded as a zero. Cells with one possible value
 * will be converted to that value.
 *
 *
 * \param cell  An encoded cell that uses bits to indicate which values could
 *              appear at this point in the board.
 * \returns     The value that must appear in the cell if there is only one
 *              possibility, or zero otherwise.
 */
__host__ __device__ int cell_to_digit(uint16_t cell) {
  // Get the index of the least-significant bit in this cell's value
#if defined(__CUDA_ARCH__)
  int msb = __clz(cell);
  int lsb = sizeof(unsigned int) * 8 - msb - 1;
#else
  int lsb = __builtin_ctz(cell);
#endif

  // Is there only one possible value for this cell? If so, return it.
  // Otherwise return zero.
  if (cell == 1 << lsb)
    return lsb;
  else
    return 0;
}

/**
 * Read in a sudoku board from a string. Boards are represented as an array of
 * 81 16-bit integers. Each integer corresponds to a cell in the board. Bits
 * 1-9 of the integer indicate whether the values 1, 2, ..., 8, or 9 could
 * appear in the given cell. A zero in the input indicates a blank cell, where
 * any value could appear.
 *
 * \param output  The location where the board will be written
 * \param str     The input string that encodes the board
 * \returns       true if parsing succeeds, false otherwise
 */
bool read_board(board_t* output, const char* str) {
  for (int index = 0; index < BOARD_DIM * BOARD_DIM; index++) {
    if (str[index] < '0' || str[index] > '9') return false;

    // Convert the character value to an equivalent integer
    int value = str[index] - '0';

    // Set the value in the board
    output->cells[index] = digit_to_cell(value);
  }

  return true;
}

/**
 * Check through a batch of boards to see how many were solved correctly.
 *
 * \param boards        An array of (hopefully) solved boards
 * \param solutions     An array of solution boards
 * \param num_boards    The number of boards and solutions
 * \param solved_count  Output: A pointer to the count of solved boards.
 * \param error:count   Output: A pointer to the count of incorrect boards.
 */
void check_solutions(board_t* boards,
                     board_t* solutions,
                     size_t num_boards,
                     size_t* solved_count,
                     size_t* error_count) {
  // Loop over all the boards in this batch
  for (int i = 0; i < num_boards; i++) {
    // Does the board match the solution?
    if (memcmp(&boards[i], &solutions[i], sizeof(board_t)) == 0) {
      // Yes. Record a solved board
      (*solved_count)++;
    } else {
      // No. Make sure the board doesn't have any constraints that rule out
      // values that are supposed to appear in the solution.
      bool valid = true;
      for (int j = 0; j < BOARD_DIM * BOARD_DIM; j++) {
        if ((boards[i].cells[j] & solutions[i].cells[j]) == 0) {
          valid = false;
        }
      }

      // If the board contains an incorrect constraint, record an error
      if (!valid) (*error_count)++;
    }
  }
}

/**
 * Entry point for the program
 */
int main(int argc, char** argv) {
  // Check arguments
  if (argc != 2) {
    fprintf(stderr, "Usage: %s <input file name>\n", argv[0]);
    exit(1);
  }

  // Try to open the input file
  FILE* input = fopen(argv[1], "r");
  if (input == NULL) {
    fprintf(stderr, "Failed to open input file %s.\n", argv[1]);
    perror(NULL);
    exit(2);
  }

  // Keep track of total boards, boards solved, and incorrect outputs
  size_t board_count = 0;
  size_t solved_count = 0;
  size_t error_count = 0;

  // Keep track of time spent solving
  size_t solving_time = 0;

  // Reserve space for a batch of boards and solutions
  board_t boards[BATCH_SIZE];
  board_t solutions[BATCH_SIZE];

  // Keep track of how many boards we've read in this batch
  size_t batch_count = 0;

  // Read the input file line-by-line
  char* line = NULL;
  size_t line_capacity = 0;
  while (getline(&line, &line_capacity, input) > 0) {
    // Read in the starting board
    if (!read_board(&boards[batch_count], line)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Read in the solution board
    if (!read_board(&solutions[batch_count], line + BOARD_DIM * BOARD_DIM + 1)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Move to the next index in the batch
    batch_count++;

    // Also increment the total count of boards
    board_count++;

    // If we finished a batch, run the solver
    if (batch_count == BATCH_SIZE) {
      size_t start_time = time_ms();
      solve_boards(boards, batch_count);
      solving_time += time_ms() - start_time;

      check_solutions(boards, solutions, batch_count, &solved_count, &error_count);

      // Reset the batch count
      batch_count = 0;
    }
  }

  // Check if there's an incomplete batch to solve
  if (batch_count > 0) {
    size_t start_time = time_ms();
    solve_boards(boards, batch_count);
    solving_time += time_ms() - start_time;

    check_solutions(boards, solutions, batch_count, &solved_count, &error_count);
  }

  // Print stats
  double seconds = (double)solving_time / 1000;
  double solving_rate = (double)solved_count / seconds;

  // Don't print nan when solver is not implemented
  if (seconds < 0.01) solving_rate = 0;

  printf("Boards: %lu\n", board_count);
  printf("Boards Solved: %lu\n", solved_count);
  printf("Errors: %lu\n", error_count);
  printf("Total Solving Time: %lums\n", solving_time);
  printf("Solving Rate: %.2f sudoku/second\n", solving_rate);

  return 0;
}
